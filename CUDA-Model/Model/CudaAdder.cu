#include "hip/hip_runtime.h"
#include "CudaAdder.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void addKernel(int* c, const int* a, const int* b, int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		c[i] = a[i] + b[i];
	}
}

/* CUDA - Compute Ability 5.3
 * This gives us ... https://en.wikipedia.org/wiki/CUDA#Version_features_and_specifications
 *
 */

std::vector<float> CudaAdder::add_matrices(const std::vector<float>& first, const std::vector<float>& second) {
	auto output = std::vector<float>(first.size());
	int* dev_a = nullptr;
	int* dev_b = nullptr;
	int* dev_c = nullptr;

	hipMalloc((void**)&dev_a, first.size() * sizeof(float));
	hipMalloc((void**)&dev_b, first.size() * sizeof(float));
	hipMalloc((void**)&dev_c, first.size() * sizeof(float));

	hipMemcpy(dev_a, first.data(), size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, second.data(), size * sizeof(float), hipMemcpyHostToDevice);

	addKernel<<<2, (first.size() + 1) / 2>>>(dev_c, dev_a, dev_b, first.size());

	hipDeviceSynchronize();

	hipMemcpy(output.data(), dev_c, first.size() * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipDeviceReset();
}
